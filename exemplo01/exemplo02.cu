/* https://devblogs.nvidia.com/even-easier-introduction-cuda/ */


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// __global__: indica que a função add deverá ser executada na
__global__
void add(int n, float *x, float *y){

	int index = threadIdx.x;
	int stride = blockDim.x;
	for(int i = index; i<n ; i=i+stride){
		y[i] = x[i]+y[i];
	}
}
int main(void){
	int N = 1<<20;
	/*
	Alocação em C++ puro
	float *x = new float[N];
	float *y = new float[N];
	*/

	/* Alocação em CUDA */

	float *x,*y;
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));

	for (int i=0;i<N;i++){
		x[i]=1.0f;
		y[i]=2.0f;
	}
	// Run kernel on 1M elements on the GPU
	//Utilizando um thread block com um 256 threads
	add<<<1,256>>>(N,x,y);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();


	float maxError = 0.0f;
	for(int i=0; i<N ; i++)
		maxError = fmax(maxError,fabs(y[i]-3.0f));
	std::cout <<"Max error: "<<maxError<<std::endl;

	
	//Free memory
	hipFree(x);
	hipFree(y);
	return 0;
}


